#include "hip/hip_runtime.h"
//
//   Copyright 2013 Pixar
//
//   Licensed under the Apache License, Version 2.0 (the "Apache License")
//   with the following modification; you may not use this file except in
//   compliance with the Apache License and the following modification to it:
//   Section 6. Trademarks. is deleted and replaced with:
//
//   6. Trademarks. This License does not grant permission to use the trade
//      names, trademarks, service marks, or product names of the Licensor
//      and its affiliates, except as required to comply with Section 4(c) of
//      the License and to reproduce the content of the NOTICE file.
//
//   You may obtain a copy of the Apache License at
//
//       http://www.apache.org/licenses/LICENSE-2.0
//
//   Unless required by applicable law or agreed to in writing, software
//   distributed under the Apache License with the above modification is
//   distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
//   KIND, either express or implied. See the Apache License for the specific
//   language governing permissions and limitations under the Apache License.
//

#include <assert.h>

template<int N> struct DeviceVertex
{
    float v[N];

    __device__ void addWithWeight(const DeviceVertex<N> *src, float weight) {
#pragma unroll
        for(int i = 0; i < N; ++i){
            v[i] += src->v[i] * weight;
        }
    }

    __device__ void clear() {
#pragma unroll
        for(int i = 0; i < N; ++i){
            v[i] = 0.0f;
        }
    }
};

// Specialize DeviceVarying for N=0 to avoid compile error:
// "flexible array member in otherwise empty struct"
template<> struct DeviceVertex<0>
{
    __device__ void addWithWeight(const DeviceVertex<0> *src, float weight) {
    }
    __device__ void clear() {
    }
};

struct DeviceTable
{
    void **tables;
    int *F0_IT;
    int *F0_ITa;
    int *E0_IT;
    int *V0_IT;
    int *V0_ITa;
    float *E0_S;
    float *V0_S;
};

__device__ void clear(float *dst, int count)
{
    for(int i = 0; i < count; ++i) dst[i] = 0;
}

__device__ void addWithWeight(float *dst, float *src, float weight, int count)
{
    for(int i = 0; i < count; ++i) dst[i] += src[i] * weight;
}

template <int NUM_VERTEX_ELEMENTS, int NUM_VARYING_ELEMENTS> __global__ void
computeFace(float *fVertex, float *fVaryings, int *F0_IT, int *F0_ITa, int offset, int tableOffset, int start, int end)
{
    DeviceVertex<NUM_VERTEX_ELEMENTS> *vertex = (DeviceVertex<NUM_VERTEX_ELEMENTS>*)fVertex;
    DeviceVertex<NUM_VARYING_ELEMENTS> *varyings = (DeviceVertex<NUM_VARYING_ELEMENTS>*)fVaryings;
    for (int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x;
         i < end + tableOffset;
         i += blockDim.x * gridDim.x) {

        int h = F0_ITa[2*i];
        int n = F0_ITa[2*i+1];
        float weight = 1.0f/n;

        DeviceVertex<NUM_VERTEX_ELEMENTS> dst;
        dst.clear();

        if(NUM_VARYING_ELEMENTS > 0){
            DeviceVertex<NUM_VARYING_ELEMENTS> dstVarying;
            dstVarying.clear();

            for(int j=0; j<n; ++j){
                int index = F0_IT[h+j];
                dst.addWithWeight(&vertex[index], weight);
                dstVarying.addWithWeight(&varyings[index], weight);
            }
            vertex[offset + i - tableOffset] = dst;
            varyings[offset + i - tableOffset] = dstVarying;
        }else{
            for(int j=0; j<n; ++j){
                int index = F0_IT[h+j];
                dst.addWithWeight(&vertex[index], weight);
            }
            vertex[offset + i - tableOffset] = dst;
        }
    }
}

__global__ void
computeFace(float *fVertex, float *fVarying,
            int vertexLength, int vertexStride,
            int varyingLength, int varyingStride,
            int *F0_IT, int *F0_ITa, int offset, int tableOffset, int start, int end)
{
    for (int i = start + tableOffset +threadIdx.x + blockIdx.x*blockDim.x;
        i < end + tableOffset;
        i += blockDim.x * gridDim.x){

        int h = F0_ITa[2*i];
        int n = F0_ITa[2*i+1];
        float weight = 1.0f/n;

        // XXX: can we use local stack like alloca?
        float *dstVertex = fVertex + (i+offset-tableOffset)*vertexStride;
        clear(dstVertex, vertexLength);
        float *dstVarying = fVarying + (i+offset-tableOffset)*varyingStride;
        clear(dstVarying, varyingLength);

        for(int j=0; j<n; ++j){
            int index = F0_IT[h+j];
            addWithWeight(dstVertex, fVertex + index*vertexStride, weight, vertexLength);
            addWithWeight(dstVarying, fVarying + index*varyingStride, weight, varyingLength);
        }
    }
}

template <int NUM_VERTEX_ELEMENTS, int NUM_VARYING_ELEMENTS> __global__ void
computeQuadFace(float *fVertex, float *fVaryings, int *F0_IT, int offset, int tableOffset, int start, int end)
{
    DeviceVertex<NUM_VERTEX_ELEMENTS> *vertex = (DeviceVertex<NUM_VERTEX_ELEMENTS>*)fVertex;
    DeviceVertex<NUM_VARYING_ELEMENTS> *varyings = (DeviceVertex<NUM_VARYING_ELEMENTS>*)fVaryings;
    for (int i = start + threadIdx.x + blockIdx.x*blockDim.x;
         i < end;
         i += blockDim.x * gridDim.x) {

        int fidx0 = F0_IT[tableOffset + 4 * i + 0];
        int fidx1 = F0_IT[tableOffset + 4 * i + 1];
        int fidx2 = F0_IT[tableOffset + 4 * i + 2];
        int fidx3 = F0_IT[tableOffset + 4 * i + 3];

        DeviceVertex<NUM_VERTEX_ELEMENTS> dst;
        dst.clear();

        if(NUM_VARYING_ELEMENTS > 0){
            DeviceVertex<NUM_VARYING_ELEMENTS> dstVarying;
            dstVarying.clear();

            dst.addWithWeight(&vertex[fidx0], 0.25f);
            dst.addWithWeight(&vertex[fidx1], 0.25f);
            dst.addWithWeight(&vertex[fidx2], 0.25f);
            dst.addWithWeight(&vertex[fidx3], 0.25f);
            dstVarying.addWithWeight(&varyings[fidx0], 0.25f);
            dstVarying.addWithWeight(&varyings[fidx1], 0.25f);
            dstVarying.addWithWeight(&varyings[fidx2], 0.25f);
            dstVarying.addWithWeight(&varyings[fidx3], 0.25f);
            vertex[offset + i] = dst;
            varyings[offset + i] = dstVarying;
        }else{
            dst.addWithWeight(&vertex[fidx0], 0.25f);
            dst.addWithWeight(&vertex[fidx1], 0.25f);
            dst.addWithWeight(&vertex[fidx2], 0.25f);
            dst.addWithWeight(&vertex[fidx3], 0.25f);
            vertex[offset + i] = dst;
        }
    }
}

__global__ void
computeQuadFace(float *fVertex, float *fVarying,
                int vertexLength, int vertexStride,
                int varyingLength, int varyingStride,
                int *F0_IT, int offset, int tableOffset, int start, int end)
{
    for (int i = start +threadIdx.x + blockIdx.x*blockDim.x;
        i < end;
        i += blockDim.x * gridDim.x){

        int fidx0 = F0_IT[tableOffset + 4 * i + 0];
        int fidx1 = F0_IT[tableOffset + 4 * i + 1];
        int fidx2 = F0_IT[tableOffset + 4 * i + 2];
        int fidx3 = F0_IT[tableOffset + 4 * i + 3];

        // XXX: can we use local stack like alloca?
        float *dstVertex = fVertex + (i+offset)*vertexStride;
        clear(dstVertex, vertexLength);
        float *dstVarying = fVarying + (i+offset)*varyingStride;
        clear(dstVarying, varyingLength);

        addWithWeight(dstVertex, fVertex + fidx0*vertexStride, 0.25f, vertexLength);
        addWithWeight(dstVertex, fVertex + fidx1*vertexStride, 0.25f, vertexLength);
        addWithWeight(dstVertex, fVertex + fidx2*vertexStride, 0.25f, vertexLength);
        addWithWeight(dstVertex, fVertex + fidx3*vertexStride, 0.25f, vertexLength);
        addWithWeight(dstVarying, fVarying + fidx0*varyingStride, 0.25f, varyingLength);
        addWithWeight(dstVarying, fVarying + fidx1*varyingStride, 0.25f, varyingLength);
        addWithWeight(dstVarying, fVarying + fidx2*varyingStride, 0.25f, varyingLength);
        addWithWeight(dstVarying, fVarying + fidx3*varyingStride, 0.25f, varyingLength);
    }
}

template <int NUM_VERTEX_ELEMENTS, int NUM_VARYING_ELEMENTS> __global__ void
computeTriQuadFace(float *fVertex, float *fVaryings, int *F0_IT, int offset, int tableOffset, int start, int end)
{
    DeviceVertex<NUM_VERTEX_ELEMENTS> *vertex = (DeviceVertex<NUM_VERTEX_ELEMENTS>*)fVertex;
    DeviceVertex<NUM_VARYING_ELEMENTS> *varyings = (DeviceVertex<NUM_VARYING_ELEMENTS>*)fVaryings;
    for (int i = start + threadIdx.x + blockIdx.x*blockDim.x;
         i < end;
         i += blockDim.x * gridDim.x) {

        int fidx0 = F0_IT[tableOffset + 4 * i + 0];
        int fidx1 = F0_IT[tableOffset + 4 * i + 1];
        int fidx2 = F0_IT[tableOffset + 4 * i + 2];
        int fidx3 = F0_IT[tableOffset + 4 * i + 3];

        bool triangle = (fidx2 == fidx3);
        float weight = triangle ? 1.0f / 3.0f : 1.0f / 4.0f;

        DeviceVertex<NUM_VERTEX_ELEMENTS> dst;
        dst.clear();

        if(NUM_VARYING_ELEMENTS > 0){
            DeviceVertex<NUM_VARYING_ELEMENTS> dstVarying;
            dstVarying.clear();

            dst.addWithWeight(&vertex[fidx0], weight);
            dst.addWithWeight(&vertex[fidx1], weight);
            dst.addWithWeight(&vertex[fidx2], weight);
            dstVarying.addWithWeight(&varyings[fidx0], weight);
            dstVarying.addWithWeight(&varyings[fidx1], weight);
            dstVarying.addWithWeight(&varyings[fidx2], weight);
            if (!triangle) {
                dst.addWithWeight(&vertex[fidx3], weight);
                dstVarying.addWithWeight(&varyings[fidx3], 0.25f);
            }
            vertex[offset + i] = dst;
            varyings[offset + i] = dstVarying;
        }else{
            dst.addWithWeight(&vertex[fidx0], weight);
            dst.addWithWeight(&vertex[fidx1], weight);
            dst.addWithWeight(&vertex[fidx2], weight);
            if (!triangle)
                dst.addWithWeight(&vertex[fidx3], weight);
            vertex[offset + i] = dst;
        }
    }
}

__global__ void
computeTriQuadFace(float *fVertex, float *fVarying,
                   int vertexLength, int vertexStride,
                   int varyingLength, int varyingStride,
                   int *F0_IT, int offset, int tableOffset, int start, int end)
{
    for (int i = start +threadIdx.x + blockIdx.x*blockDim.x;
        i < end;
        i += blockDim.x * gridDim.x){

        int fidx0 = F0_IT[tableOffset + 4 * i + 0];
        int fidx1 = F0_IT[tableOffset + 4 * i + 1];
        int fidx2 = F0_IT[tableOffset + 4 * i + 2];
        int fidx3 = F0_IT[tableOffset + 4 * i + 3];

        bool triangle = (fidx2 == fidx3);
        float weight = triangle ? 1.0f / 3.0f : 1.0f / 4.0f;

        // XXX: can we use local stack like alloca?
        float *dstVertex = fVertex + (i+offset)*vertexStride;
        clear(dstVertex, vertexLength);
        float *dstVarying = fVarying + (i+offset)*varyingStride;
        clear(dstVarying, varyingLength);

        addWithWeight(dstVertex, fVertex + fidx0*vertexStride, weight, vertexLength);
        addWithWeight(dstVertex, fVertex + fidx1*vertexStride, weight, vertexLength);
        addWithWeight(dstVertex, fVertex + fidx2*vertexStride, weight, vertexLength);
        addWithWeight(dstVarying, fVarying + fidx0*varyingStride, weight, varyingLength);
        addWithWeight(dstVarying, fVarying + fidx1*varyingStride, weight, varyingLength);
        addWithWeight(dstVarying, fVarying + fidx2*varyingStride, weight, varyingLength);
        if (!triangle) {
            addWithWeight(dstVertex, fVertex + fidx3*vertexStride, weight, vertexLength);
            addWithWeight(dstVarying, fVarying + fidx3*varyingStride, weight, varyingLength);
        }
    }
}

template <int NUM_VERTEX_ELEMENTS, int NUM_VARYING_ELEMENTS> __global__ void
computeEdge(float *fVertex, float *fVaryings, int *E0_IT, float *E0_S, int offset, int tableOffset, int start, int end)
{
    DeviceVertex<NUM_VERTEX_ELEMENTS> *vertex = (DeviceVertex<NUM_VERTEX_ELEMENTS>*)fVertex;
    DeviceVertex<NUM_VARYING_ELEMENTS> *varyings = (DeviceVertex<NUM_VARYING_ELEMENTS>*)fVaryings;

    for (int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x;
         i < end + tableOffset;
         i+= blockDim.x * gridDim.x){

        int eidx0 = E0_IT[4*i+0];
        int eidx1 = E0_IT[4*i+1];
        int eidx2 = E0_IT[4*i+2];
        int eidx3 = E0_IT[4*i+3];

        float vertWeight = E0_S[i*2+0];

        // Fully sharp edge : vertWeight = 0.5f;
        DeviceVertex<NUM_VERTEX_ELEMENTS> dst;
        dst.clear();

        dst.addWithWeight(&vertex[eidx0], vertWeight);
        dst.addWithWeight(&vertex[eidx1], vertWeight);

        if(eidx2 > -1){
            float faceWeight = E0_S[i*2+1];

            dst.addWithWeight(&vertex[eidx2], faceWeight);
            dst.addWithWeight(&vertex[eidx3], faceWeight);
        }
        vertex[offset+i-tableOffset] = dst;

        if(NUM_VARYING_ELEMENTS > 0){
            DeviceVertex<NUM_VARYING_ELEMENTS> dstVarying;
            dstVarying.clear();
            dstVarying.addWithWeight(&varyings[eidx0], 0.5f);
            dstVarying.addWithWeight(&varyings[eidx1], 0.5f);
            varyings[offset+i-tableOffset] = dstVarying;
        }
    }
}

__global__ void
computeEdge(float *fVertex, float *fVarying,
            int vertexLength, int vertexStride,
            int varyingLength, int varyingStride,
            int *E0_IT, float *E0_S, int offset, int tableOffset, int start, int end)
{
    for (int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x;
         i < end + tableOffset;i+= blockDim.x * gridDim.x) {

        int eidx0 = E0_IT[4*i+0];
        int eidx1 = E0_IT[4*i+1];
        int eidx2 = E0_IT[4*i+2];
        int eidx3 = E0_IT[4*i+3];

        float vertWeight = E0_S[i*2+0];

        // Fully sharp edge : vertWeight = 0.5f;
        float *dstVertex = fVertex + (i+offset-tableOffset)*vertexStride;
        clear(dstVertex, vertexLength);

        addWithWeight(dstVertex, fVertex + eidx0*vertexStride, vertWeight, vertexLength);
        addWithWeight(dstVertex, fVertex + eidx1*vertexStride, vertWeight, vertexLength);

        if(eidx2 > -1){
            float faceWeight = E0_S[i*2+1];

            addWithWeight(dstVertex, fVertex + eidx2*vertexStride, faceWeight, vertexLength);
            addWithWeight(dstVertex, fVertex + eidx3*vertexStride, faceWeight, vertexLength);
        }

        if (varyingLength > 0){
            float *dstVarying = fVarying + (i+offset-tableOffset)*varyingStride;
            clear(dstVarying, varyingLength);

            addWithWeight(dstVarying, fVarying + eidx0*varyingStride, 0.5f, varyingLength);
            addWithWeight(dstVarying, fVarying + eidx1*varyingStride, 0.5f, varyingLength);
        }
    }
}

template <int NUM_VERTEX_ELEMENTS, int NUM_VARYING_ELEMENTS> __global__ void
computeVertexA(float *fVertex, float *fVaryings, int *V0_ITa, float *V0_S, int offset, int tableOffset, int start, int end, int pass)
{
    DeviceVertex<NUM_VERTEX_ELEMENTS> *vertex = (DeviceVertex<NUM_VERTEX_ELEMENTS>*)fVertex;
    DeviceVertex<NUM_VARYING_ELEMENTS> *varyings = (DeviceVertex<NUM_VARYING_ELEMENTS>*)fVaryings;
    for (int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x;
         i < end+tableOffset;
         i += blockDim.x * gridDim.x) {

        int n     = V0_ITa[5*i+1];
        int p     = V0_ITa[5*i+2];
        int eidx0 = V0_ITa[5*i+3];
        int eidx1 = V0_ITa[5*i+4];

        float weight = (pass==1) ? V0_S[i] : 1.0f - V0_S[i];

        // In the case of fractional weight, the weight must be inverted since
        // the value is shared with the k_Smooth kernel (statistically the
        // k_Smooth kernel runs much more often than this one)
        if (weight>0.0f && weight<1.0f && n > 0)
            weight=1.0f-weight;

        DeviceVertex<NUM_VERTEX_ELEMENTS> dst;
        if (not pass) {
            dst.clear();
        } else {
            dst = vertex[i+offset-tableOffset];
        }

        if (eidx0==-1 || (pass==0 && (n==-1)) ) {
            dst.addWithWeight(&vertex[p], weight);
        } else {
            dst.addWithWeight(&vertex[p], weight * 0.75f);
            dst.addWithWeight(&vertex[eidx0], weight * 0.125f);
            dst.addWithWeight(&vertex[eidx1], weight * 0.125f);
        }
        vertex[i+offset-tableOffset] = dst;

        if(NUM_VARYING_ELEMENTS > 0){
            if(not pass){
                DeviceVertex<NUM_VARYING_ELEMENTS> dstVarying;
                dstVarying.clear();
                dstVarying.addWithWeight(&varyings[p], 1.0f);
                varyings[i+offset-tableOffset] = dstVarying;
            }
        }
    }
}

__global__ void
computeVertexA(float *fVertex, float *fVaryings,
               int vertexLength, int vertexStride,
               int varyingLength, int varyingStride,
               int *V0_ITa, float *V0_S, int offset, int tableOffset, int start, int end, int pass)
{
    for (int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x;
         i < end + tableOffset;
         i += blockDim.x * gridDim.x){

        int n     = V0_ITa[5*i+1];
        int p     = V0_ITa[5*i+2];
        int eidx0 = V0_ITa[5*i+3];
        int eidx1 = V0_ITa[5*i+4];

        float weight = (pass==1) ? V0_S[i] : 1.0f - V0_S[i];

        // In the case of fractional weight, the weight must be inverted since
        // the value is shared with the k_Smooth kernel (statistically the
        // k_Smooth kernel runs much more often than this one)
        if (weight>0.0f && weight<1.0f && n > 0)
            weight=1.0f-weight;

        float *dstVertex = fVertex + (i+offset-tableOffset)*vertexStride;
        if (not pass) {
            clear(dstVertex, vertexLength);
        }

        if (eidx0==-1 || (pass==0 && (n==-1)) ) {
            addWithWeight(dstVertex, fVertex + p*vertexStride, weight, vertexLength);
        } else {
            addWithWeight(dstVertex, fVertex + p*vertexStride, weight*0.75f, vertexLength);
            addWithWeight(dstVertex, fVertex + eidx0*vertexStride, weight*0.125f, vertexLength);
            addWithWeight(dstVertex, fVertex + eidx1*vertexStride, weight*0.125f, vertexLength);
        }

        if(varyingLength > 0){
            if(not pass){
                float *dstVarying = fVaryings + (i+offset-tableOffset)*varyingStride;
                clear(dstVarying, varyingLength);
                addWithWeight(dstVarying, fVaryings + p*varyingStride, 1.0f, varyingLength);
            }
        }
    }

}


//texture <int, 1> texV0_IT;

template <int NUM_VERTEX_ELEMENTS, int NUM_VARYING_ELEMENTS> __global__ void
computeVertexB(float *fVertex, float *fVaryings,
               const int *V0_ITa, const int *V0_IT, const float *V0_S, int offset, int tableOffset, int start, int end)
{
    DeviceVertex<NUM_VERTEX_ELEMENTS> *vertex = (DeviceVertex<NUM_VERTEX_ELEMENTS>*)fVertex;
    DeviceVertex<NUM_VARYING_ELEMENTS> *varyings = (DeviceVertex<NUM_VARYING_ELEMENTS>*)fVaryings;
    for (int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x;
         i < end + tableOffset;
         i += blockDim.x * gridDim.x) {

        int h = V0_ITa[5*i];
        int n = V0_ITa[5*i+1];
        int p = V0_ITa[5*i+2];

        float weight = V0_S[i];
        float wp = 1.0f/float(n*n);
        float wv = (n-2.0f) * n * wp;

        DeviceVertex<NUM_VERTEX_ELEMENTS> dst;
        dst.clear();
        dst.addWithWeight(&vertex[p], weight * wv);

        for (int j = 0; j < n; ++j) {
            dst.addWithWeight(&vertex[V0_IT[h+j*2]], weight * wp);
            dst.addWithWeight(&vertex[V0_IT[h+j*2+1]], weight * wp);
//            int idx0 = tex1Dfetch(texV0_IT, h+j*2);
//            int idx1 = tex1Dfetch(texV0_IT, h+j*2+1);
//            dst.addWithWeight(&vertex[idx0], weight * wp);
//            dst.addWithWeight(&vertex[idx1], weight * wp);
        }
        vertex[i+offset-tableOffset] = dst;

        if(NUM_VARYING_ELEMENTS > 0){
            DeviceVertex<NUM_VARYING_ELEMENTS> dstVarying;
            dstVarying.clear();
            dstVarying.addWithWeight(&varyings[p], 1.0f);
            varyings[i+offset-tableOffset] = dstVarying;
        }
    }
}

__global__ void
computeVertexB(float *fVertex, float *fVarying,
               int vertexLength, int vertexStride,
               int varyingLength, int varyingStride,
               const int *V0_ITa, const int *V0_IT, const float *V0_S, int offset, int tableOffset, int start, int end)
{
    for (int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x;
         i < end + tableOffset;
         i += blockDim.x * gridDim.x) {

        int h = V0_ITa[5*i];
        int n = V0_ITa[5*i+1];
        int p = V0_ITa[5*i+2];

        float weight = V0_S[i];
        float wp = 1.0f/float(n*n);
        float wv = (n-2.0f) * n * wp;

        float *dstVertex = fVertex + (i+offset-tableOffset)*vertexStride;
        clear(dstVertex, vertexLength);
        addWithWeight(dstVertex, fVertex + p*vertexStride, weight*wv, vertexLength);

        for (int j = 0; j < n; ++j) {
            addWithWeight(dstVertex, fVertex + V0_IT[h+j*2]*vertexStride, weight*wp, vertexLength);
            addWithWeight(dstVertex, fVertex + V0_IT[h+j*2+1]*vertexStride, weight*wp, vertexLength);
        }

        if (varyingLength > 0) {
            float *dstVarying = fVarying + (i+offset-tableOffset)*varyingStride;
            clear(dstVarying, varyingLength);
            addWithWeight(dstVarying, fVarying + p*varyingStride, 1.0f, varyingLength);
        }
    }
}


// --------------------------------------------------------------------------------------------

template <int NUM_VERTEX_ELEMENTS, int NUM_VARYING_ELEMENTS> __global__ void
computeLoopVertexB(float *fVertex, float *fVaryings, int *V0_ITa, int *V0_IT, float *V0_S, int offset, int tableOffset, int start, int end)
{
    DeviceVertex<NUM_VERTEX_ELEMENTS> *vertex = (DeviceVertex<NUM_VERTEX_ELEMENTS>*)fVertex;
    DeviceVertex<NUM_VARYING_ELEMENTS> *varyings = (DeviceVertex<NUM_VARYING_ELEMENTS>*)fVaryings;
    for (int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x;
         i < end + tableOffset;
         i += blockDim.x * gridDim.x) {

        int h = V0_ITa[5*i];
        int n = V0_ITa[5*i+1];
        int p = V0_ITa[5*i+2];

        float weight = V0_S[i];
        float wp = 1.0f/float(n);
        float beta = 0.25f * __cosf(float(M_PI) * 2.0f * wp) + 0.375f;
        beta = beta * beta;
        beta = (0.625f - beta) * wp;

        DeviceVertex<NUM_VERTEX_ELEMENTS> dst;
        dst.clear();

        dst.addWithWeight(&vertex[p], weight * (1.0f - (beta * n)));

        for (int j = 0; j < n; ++j) {
            dst.addWithWeight(&vertex[V0_IT[h+j]], weight * beta);
        }
        vertex[i+offset-tableOffset] = dst;

        if (NUM_VARYING_ELEMENTS > 0) {
            DeviceVertex<NUM_VARYING_ELEMENTS> dstVarying;
            dstVarying.clear();
            dstVarying.addWithWeight(&varyings[p], 1.0f);
            varyings[i+offset-tableOffset] = dstVarying;
        }
    }
}

__global__ void
computeLoopVertexB(float *fVertex, float *fVarying,
                   int vertexLength, int vertexStride,
                   int varyingLength, int varyingStride,
                   const int *V0_ITa, const int *V0_IT, const float *V0_S, int offset, int tableOffset, int start, int end)
{
    for (int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x;
         i < end + tableOffset;
         i += blockDim.x * gridDim.x) {

        int h = V0_ITa[5*i];
        int n = V0_ITa[5*i+1];
        int p = V0_ITa[5*i+2];

        float weight = V0_S[i];
        float wp = 1.0f/float(n);
        float beta = 0.25f * __cosf(float(M_PI) * 2.0f * wp) + 0.375f;
        beta = beta * beta;
        beta = (0.625f - beta) * wp;

        float *dstVertex = fVertex + (i+offset-tableOffset)*vertexStride;
        clear(dstVertex, vertexLength);
        addWithWeight(dstVertex, fVertex + p*vertexStride, weight*(1.0f-(beta*n)), vertexLength);

        for (int j = 0; j < n; ++j) {
            addWithWeight(dstVertex, fVertex + V0_IT[h+j]*vertexStride, weight*beta, vertexLength);
        }

        if (varyingLength > 0) {
            float *dstVarying = fVarying + (i+offset-tableOffset)*varyingStride;
            clear(dstVarying, varyingLength);
            addWithWeight(dstVarying, fVarying + p*varyingStride, 1.0f, varyingLength);
        }
    }
}

// --------------------------------------------------------------------------------------------

template <int NUM_VERTEX_ELEMENTS, int NUM_VARYING_ELEMENTS> __global__ void
computeBilinearEdge(float *fVertex, float *fVaryings, int *E0_IT, int offset, int tableOffset, int start, int end)
{
    DeviceVertex<NUM_VERTEX_ELEMENTS> *vertex = (DeviceVertex<NUM_VERTEX_ELEMENTS>*)fVertex;
    DeviceVertex<NUM_VARYING_ELEMENTS> *varyings = (DeviceVertex<NUM_VARYING_ELEMENTS>*)fVaryings;
    for (int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x;
         i < end + tableOffset;
         i+= blockDim.x * gridDim.x) {

        int eidx0 = E0_IT[2*i+0];
        int eidx1 = E0_IT[2*i+1];

        DeviceVertex<NUM_VERTEX_ELEMENTS> dst;
        dst.clear();

        dst.addWithWeight(&vertex[eidx0], 0.5f);
        dst.addWithWeight(&vertex[eidx1], 0.5f);

        vertex[offset+i-tableOffset] = dst;

        if (NUM_VARYING_ELEMENTS > 0) {
            DeviceVertex<NUM_VARYING_ELEMENTS> dstVarying;
            dstVarying.clear();
            dstVarying.addWithWeight(&varyings[eidx0], 0.5f);
            dstVarying.addWithWeight(&varyings[eidx1], 0.5f);
            varyings[offset+i-tableOffset] = dstVarying;
        }
    }
}

__global__ void
computeBilinearEdge(float *fVertex, float *fVarying,
                    int vertexLength, int vertexStride,
                    int varyingLength, int varyingStride,
                    int *E0_IT, int offset, int tableOffset, int start, int end)
{
    for (int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x;
         i < end + tableOffset;
         i+= blockDim.x * gridDim.x) {

        int eidx0 = E0_IT[2*i+0];
        int eidx1 = E0_IT[2*i+1];

        float *dstVertex = fVertex + (i+offset-tableOffset)*vertexStride;
        clear(dstVertex, vertexLength);

        addWithWeight(dstVertex, fVertex + eidx0*vertexStride, 0.5f, vertexLength);
        addWithWeight(dstVertex, fVertex + eidx1*vertexStride, 0.5f, vertexLength);

        if (varyingLength > 0) {
            float *dstVarying = fVarying + (i+offset-tableOffset)*varyingStride;
            clear(dstVarying, varyingLength);

            addWithWeight(dstVarying, fVarying + eidx0*varyingStride, 0.5f, varyingLength);
            addWithWeight(dstVarying, fVarying + eidx1*varyingStride, 0.5f, varyingLength);
        }
    }
}

template <int NUM_VERTEX_ELEMENTS, int NUM_VARYING_ELEMENTS> __global__ void
computeBilinearVertex(float *fVertex, float *fVaryings, int *V0_ITa, int offset, int tableOffset, int start, int end)
{
    DeviceVertex<NUM_VERTEX_ELEMENTS> *vertex = (DeviceVertex<NUM_VERTEX_ELEMENTS>*)fVertex;
    DeviceVertex<NUM_VARYING_ELEMENTS> *varyings = (DeviceVertex<NUM_VARYING_ELEMENTS>*)fVaryings;
    for (int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x;
         i < end + tableOffset;
         i += blockDim.x * gridDim.x) {

        int p = V0_ITa[i];

        DeviceVertex<NUM_VERTEX_ELEMENTS> dst;
        dst.clear();

        dst.addWithWeight(&vertex[p], 1.0f);
        vertex[i+offset-tableOffset] = dst;

        if (NUM_VARYING_ELEMENTS > 0) {
            DeviceVertex<NUM_VARYING_ELEMENTS> dstVarying;
            dstVarying.clear();
            dstVarying.addWithWeight(&varyings[p], 1.0f);
            varyings[i+offset-tableOffset] = dstVarying;
        }
    }
}

__global__ void
computeBilinearVertex(float *fVertex, float *fVarying,
                      int vertexLength, int vertexStride,
                      int varyingLength, int varyingStride,
                      const int *V0_ITa, int offset, int tableOffset, int start, int end)
{
    for (int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x;
         i < end + tableOffset;
         i += blockDim.x * gridDim.x) {

        int p = V0_ITa[i];

        float *dstVertex = fVertex + (i+offset-tableOffset)*vertexStride;
        clear(dstVertex, vertexLength);
        addWithWeight(dstVertex, fVertex + p*vertexStride, 1.0f, vertexLength);

        if (varyingLength > 0) {
            float *dstVarying = fVarying + (i+offset-tableOffset)*varyingStride;
            clear(dstVarying, varyingLength);
            addWithWeight(dstVarying, fVarying + p*varyingStride, 1.0f, varyingLength);
        }
    }
}

// --------------------------------------------------------------------------------------------

__global__ void
editVertexAdd(float *fVertex, int vertexLength, int vertexStride,
              int primVarOffset, int primVarWidth,
              int vertexOffset, int tableOffset, int start, int end,
              const int *editIndices, const float *editValues)
{
    for (int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x;
         i < end + tableOffset;
         i += blockDim.x * gridDim.x) {

        float *dstVertex = fVertex + (editIndices[i] + vertexOffset) * vertexStride + primVarOffset;

        for(int j = 0; j < primVarWidth; j++) {
            *dstVertex++ += editValues[i*primVarWidth + j];
        }
    }
}

// --------------------------------------------------------------------------------------------

#include "../version.h"

// XXX: this macro usage is tentative. Since cuda kernel can't be dynamically configured,
// still trying to find better way to have optimized kernel..

#define OPT_KERNEL(NUM_VERTEX_ELEMENTS, NUM_VARYING_ELEMENTS, KERNEL, X, Y, STREAM, ARG) \
    if(vertexLength == NUM_VERTEX_ELEMENTS &&                           \
       varyingLength == NUM_VARYING_ELEMENTS &&                         \
       vertexStride == vertexLength &&                                  \
       varyingStride == varyingLength)                                  \
    { KERNEL<NUM_VERTEX_ELEMENTS, NUM_VARYING_ELEMENTS><<<X,Y,0,STREAM>>>ARG;    \
        return;  }

extern "C" {

void OsdCudaComputeFace(hipStream_t stream,
                        float *vertex, float *varying,
                        int vertexLength, int vertexStride,
                        int varyingLength, int varyingStride,
                        int *F_IT, int *F_ITa, int offset, int tableOffset, int start, int end)
{
    //computeFace<3, 0><<<512,32,0,stream>>>(vertex, varying, F_IT, F_ITa, offset, start, end);
    OPT_KERNEL(0, 0, computeFace, 512, 32, stream, (vertex, varying, F_IT, F_ITa, offset, tableOffset, start, end));
    OPT_KERNEL(0, 3, computeFace, 512, 32, stream, (vertex, varying, F_IT, F_ITa, offset, tableOffset, start, end));
    OPT_KERNEL(3, 0, computeFace, 512, 32, stream, (vertex, varying, F_IT, F_ITa, offset, tableOffset, start, end));
    OPT_KERNEL(3, 3, computeFace, 512, 32, stream, (vertex, varying, F_IT, F_ITa, offset, tableOffset, start, end));

    // fallback kernel (slow)
    computeFace<<<512, 32, 0, stream>>>(vertex, varying,
                             vertexLength, vertexStride, varyingLength, varyingStride,
                             F_IT, F_ITa, offset, tableOffset, start, end);
}

void OsdCudaComputeQuadFace(hipStream_t stream,
                            float *vertex, float *varying,
                            int vertexLength, int vertexStride,
                            int varyingLength, int varyingStride,
                            int *F_IT, int offset, int tableOffset, int start, int end)
{
    //computeQuadFace<3, 0><<<512,32,0,stream>>>(vertex, varying, F_IT, offset, start, end);
    OPT_KERNEL(0, 0, computeQuadFace, 512, 32, stream, (vertex, varying, F_IT, offset, tableOffset, start, end));
    OPT_KERNEL(0, 3, computeQuadFace, 512, 32, stream, (vertex, varying, F_IT, offset, tableOffset, start, end));
    OPT_KERNEL(3, 0, computeQuadFace, 512, 32, stream, (vertex, varying, F_IT, offset, tableOffset, start, end));
    OPT_KERNEL(3, 3, computeQuadFace, 512, 32, stream, (vertex, varying, F_IT, offset, tableOffset, start, end));

    // fallback kernel (slow)
    computeQuadFace<<<512, 32, 0, stream>>>(vertex, varying,
                                 vertexLength, vertexStride, varyingLength, varyingStride,
                                 F_IT, offset, tableOffset, start, end);
}

void OsdCudaComputeTriQuadFace(hipStream_t stream,
                               float *vertex, float *varying,
                               int vertexLength, int vertexStride,
                               int varyingLength, int varyingStride,
                               int *F_IT, int offset, int tableOffset, int start, int end)
{
    //computeTriQuadFace<3, 0><<<512,32,0,stream>>>(vertex, varying, F_IT, offset, start, end);
    OPT_KERNEL(0, 0, computeTriQuadFace, 512, 32, stream, (vertex, varying, F_IT, offset, tableOffset, start, end));
    OPT_KERNEL(0, 3, computeTriQuadFace, 512, 32, stream, (vertex, varying, F_IT, offset, tableOffset, start, end));
    OPT_KERNEL(3, 0, computeTriQuadFace, 512, 32, stream, (vertex, varying, F_IT, offset, tableOffset, start, end));
    OPT_KERNEL(3, 3, computeTriQuadFace, 512, 32, stream, (vertex, varying, F_IT, offset, tableOffset, start, end));

    // fallback kernel (slow)
    computeTriQuadFace<<<512, 32, 0, stream>>>(vertex, varying,
                                    vertexLength, vertexStride, varyingLength, varyingStride,
                                    F_IT, offset, tableOffset, start, end);
}


void OsdCudaComputeEdge(hipStream_t stream,
                        float *vertex, float *varying,
                        int vertexLength, int vertexStride,
                        int varyingLength, int varyingStride,
                        int *E_IT, float *E_W, int offset, int tableOffset, int start, int end)
{
    //computeEdge<0, 3><<<512,32,0,stream>>>(vertex, varying, E_IT, E_W, offset, start, end);
    OPT_KERNEL(0, 0, computeEdge, 512, 32, stream, (vertex, varying, E_IT, E_W, offset, tableOffset, start, end));
    OPT_KERNEL(0, 3, computeEdge, 512, 32, stream, (vertex, varying, E_IT, E_W, offset, tableOffset, start, end));
    OPT_KERNEL(3, 0, computeEdge, 512, 32, stream, (vertex, varying, E_IT, E_W, offset, tableOffset, start, end));
    OPT_KERNEL(3, 3, computeEdge, 512, 32, stream, (vertex, varying, E_IT, E_W, offset, tableOffset, start, end));

    // fallback kernel (slow)
    computeEdge<<<512, 32, 0, stream>>>(vertex, varying,
                             vertexLength, vertexStride, varyingLength, varyingStride,
                             E_IT, E_W, offset, tableOffset, start, end);
}

void OsdCudaComputeVertexA(hipStream_t stream,
                           float *vertex, float *varying,
                           int vertexLength, int vertexStride,
                           int varyingLength, int varyingStride,
                           int *V_ITa, float *V_W, int offset, int tableOffset, int start, int end, int pass)
{
//    computeVertexA<0, 3><<<512,32,0,stream>>>(vertex, varying, V_ITa, V_W, offset, start, end, pass);
    OPT_KERNEL(0, 0, computeVertexA, 512, 32, stream, (vertex, varying, V_ITa, V_W, offset, tableOffset, start, end, pass));
    OPT_KERNEL(0, 3, computeVertexA, 512, 32, stream, (vertex, varying, V_ITa, V_W, offset, tableOffset, start, end, pass));
    OPT_KERNEL(3, 0, computeVertexA, 512, 32, stream, (vertex, varying, V_ITa, V_W, offset, tableOffset, start, end, pass));
    OPT_KERNEL(3, 3, computeVertexA, 512, 32, stream, (vertex, varying, V_ITa, V_W, offset, tableOffset, start, end, pass));

    // fallback kernel (slow)
    computeVertexA<<<512, 32, 0, stream>>>(vertex, varying,
                                vertexLength, vertexStride, varyingLength, varyingStride,
                                V_ITa, V_W, offset, tableOffset, start, end, pass);
}

void OsdCudaComputeVertexB(hipStream_t stream,
                           float *vertex, float *varying,
                           int vertexLength, int vertexStride,
                           int varyingLength, int varyingStride,
                           int *V_ITa, int *V_IT, float *V_W, int offset, int tableOffset, int start, int end)
{
//    computeVertexB<0, 3><<<512,32,0,stream>>>(vertex, varying, V_ITa, V_IT, V_W, offset, start, end);
    OPT_KERNEL(0, 0, computeVertexB, 512, 32, stream, (vertex, varying, V_ITa, V_IT, V_W, offset, tableOffset, start, end));
    OPT_KERNEL(0, 3, computeVertexB, 512, 32, stream, (vertex, varying, V_ITa, V_IT, V_W, offset, tableOffset, start, end));
    OPT_KERNEL(3, 0, computeVertexB, 512, 32, stream, (vertex, varying, V_ITa, V_IT, V_W, offset, tableOffset, start, end));
    OPT_KERNEL(3, 3, computeVertexB, 512, 32, stream, (vertex, varying, V_ITa, V_IT, V_W, offset, tableOffset, start, end));

    // fallback kernel (slow)
    computeVertexB<<<512, 32, 0, stream>>>(vertex, varying,
                                vertexLength, vertexStride, varyingLength, varyingStride,
                                V_ITa, V_IT, V_W, offset, tableOffset, start, end);
}

void OsdCudaComputeLoopVertexB(hipStream_t stream,
                               float *vertex, float *varying,
                               int vertexLength, int vertexStride,
                               int varyingLength, int varyingStride,
                               int *V_ITa, int *V_IT, float *V_W, int offset, int tableOffset, int start, int end)
{
//    computeLoopVertexB<0, 3><<<512,32,0,stream>>>(vertex, varying, V_ITa, V_IT, V_W, offset, start, end);
    OPT_KERNEL(0, 0, computeLoopVertexB, 512, 32, stream, (vertex, varying, V_ITa, V_IT, V_W, offset, tableOffset, start, end));
    OPT_KERNEL(0, 3, computeLoopVertexB, 512, 32, stream, (vertex, varying, V_ITa, V_IT, V_W, offset, tableOffset, start, end));
    OPT_KERNEL(3, 0, computeLoopVertexB, 512, 32, stream, (vertex, varying, V_ITa, V_IT, V_W, offset, tableOffset, start, end));
    OPT_KERNEL(3, 3, computeLoopVertexB, 512, 32, stream, (vertex, varying, V_ITa, V_IT, V_W, offset, tableOffset, start, end));

    // fallback kernel (slow)
    computeLoopVertexB<<<512, 32, 0, stream>>>(vertex, varying,
                                    vertexLength, vertexStride, varyingLength, varyingStride,
                                    V_ITa, V_IT, V_W, offset, tableOffset, start, end);
}

void OsdCudaComputeBilinearEdge(hipStream_t stream,
                                float *vertex, float *varying,
                                int vertexLength, int vertexStride,
                                int varyingLength, int varyingStride,
                                int *E_IT, int offset, int tableOffset, int start, int end)
{
    //computeBilinearEdge<0, 3><<<512,32,0,stream>>>(vertex, varying, E_IT, offset, start, end);
    OPT_KERNEL(0, 0, computeBilinearEdge, 512, 32, stream, (vertex, varying, E_IT, offset, tableOffset, start, end));
    OPT_KERNEL(0, 3, computeBilinearEdge, 512, 32, stream, (vertex, varying, E_IT, offset, tableOffset, start, end));
    OPT_KERNEL(3, 0, computeBilinearEdge, 512, 32, stream, (vertex, varying, E_IT, offset, tableOffset, start, end));
    OPT_KERNEL(3, 3, computeBilinearEdge, 512, 32, stream, (vertex, varying, E_IT, offset, tableOffset, start, end));

    // fallback kernel (slow)
    computeBilinearEdge<<<512, 32, 0, stream>>>(vertex, varying,
                                     vertexLength, vertexStride, varyingLength, varyingStride,
                                     E_IT, offset, tableOffset, start, end);
}

void OsdCudaComputeBilinearVertex(hipStream_t stream,
                                  float *vertex, float *varying,
                                  int vertexLength, int vertexStride,
                                  int varyingLength, int varyingStride,
                                  int *V_ITa, int offset, int tableOffset, int start, int end)
{
//    computeBilinearVertex<0, 3><<<512,32,0,stream>>>(vertex, varying, V_ITa, offset, start, end);
    OPT_KERNEL(0, 0, computeBilinearVertex, 512, 32, stream, (vertex, varying, V_ITa, offset, tableOffset, start, end));
    OPT_KERNEL(0, 3, computeBilinearVertex, 512, 32, stream, (vertex, varying, V_ITa, offset, tableOffset, start, end));
    OPT_KERNEL(3, 0, computeBilinearVertex, 512, 32, stream, (vertex, varying, V_ITa, offset, tableOffset, start, end));
    OPT_KERNEL(3, 3, computeBilinearVertex, 512, 32, stream, (vertex, varying, V_ITa, offset, tableOffset, start, end));

    // fallback kernel (slow)
    computeBilinearVertex<<<512, 32, 0, stream>>>(vertex, varying,
                                       vertexLength, vertexStride, varyingLength, varyingStride,
                                       V_ITa, offset, tableOffset, start, end);
}

void OsdCudaEditVertexAdd(hipStream_t stream,
                          float *vertex, int vertexLength, int vertexStride,
                          int primVarOffset, int primVarWidth,
                          int vertexOffset, int tableOffset,
                          int start, int end, int *editIndices, float *editValues)
{
    editVertexAdd<<<512, 32, 0, stream>>>(vertex, vertexLength, vertexStride, primVarOffset, primVarWidth,
                               vertexOffset, tableOffset, start, end,
                               editIndices, editValues);
}

}  /* extern "C" */
