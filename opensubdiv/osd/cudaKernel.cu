#include "hip/hip_runtime.h"
//
//     Copyright 2013 Pixar
//
//     Licensed under the Apache License, Version 2.0 (the "License");
//     you may not use this file except in compliance with the License
//     and the following modification to it: Section 6 Trademarks.
//     deleted and replaced with:
//
//     6. Trademarks. This License does not grant permission to use the
//     trade names, trademarks, service marks, or product names of the
//     Licensor and its affiliates, except as required for reproducing
//     the content of the NOTICE file.
//
//     You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
//     Unless required by applicable law or agreed to in writing,
//     software distributed under the License is distributed on an
//     "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND,
//     either express or implied.  See the License for the specific
//     language governing permissions and limitations under the
//     License.
//

#include <assert.h>

template<int N> struct DeviceVertex
{
    float pos[3];
    float userVertexData[N];

    __device__ void addWithWeight(const DeviceVertex<N> *src, float weight) {
        pos[0] += src->pos[0] * weight;
        pos[1] += src->pos[1] * weight;
        pos[2] += src->pos[2] * weight;

        for(int i = 0; i < N; ++i){
            userVertexData[i] += src->userVertexData[i] * weight;
        }
    }
    __device__ void clear() {
        pos[0] = pos[1] = pos[2] = 0.0f;
        for(int i = 0; i < N; ++i){
            userVertexData[i] = 0.0f;
        }
    }
};

template<int N> struct DeviceVarying
{
    float v[N];

    __device__ void addVaryingWithWeight(const DeviceVarying<N> *src, float weight) {
        for(int i = 0; i < N; ++i){
            v[i] += src->v[i] * weight;
        }
    }
    __device__ void clear() {
        for(int i = 0; i < N; ++i){
            v[i] = 0.0f;
        }
    }
};

// Specialize DeviceVarying for N=0 to avoid compile error:
// "flexible array member in otherwise empty struct"
template<> struct DeviceVarying<0>
{
    __device__ void addVaryingWithWeight(const DeviceVarying<0> *src, float weight) {
    }
    __device__ void clear() {
    }
};

struct DeviceTable
{
    void **tables;
    int *F0_IT;
    int *F0_ITa;
    int *E0_IT;
    int *V0_IT;
    int *V0_ITa;
    float *E0_S;
    float *V0_S;
};

__device__ void clear(float *dst, int count)
{
    for(int i = 0; i < count; ++i) dst[i] = 0;
}

__device__ void addWithWeight(float *dst, float *src, float weight, int count)
{
    for(int i = 0; i < count; ++i) dst[i] += src[i] * weight;
}

__device__ void addVaryingWithWeight(float *dst, float *src, float weight, int count)
{
    for(int i = 0; i < count; ++i) dst[i] += src[i] * weight;
}

template <int NUM_USER_VERTEX_ELEMENTS, int NUM_VARYING_ELEMENTS> __global__ void
computeFace(float *fVertex, float *fVaryings, int *F0_IT, int *F0_ITa, int offset, int tableOffset, int start, int end)
{
    DeviceVertex<NUM_USER_VERTEX_ELEMENTS> *vertex = (DeviceVertex<NUM_USER_VERTEX_ELEMENTS>*)fVertex;
    DeviceVarying<NUM_VARYING_ELEMENTS> *varyings = (DeviceVarying<NUM_VARYING_ELEMENTS>*)fVaryings;
    for(int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x; i < end + tableOffset; i += blockDim.x * gridDim.x){
        int h = F0_ITa[2*i];
        int n = F0_ITa[2*i+1];
        float weight = 1.0f/n;

        DeviceVertex<NUM_USER_VERTEX_ELEMENTS> dst;
        dst.clear();

        if(NUM_VARYING_ELEMENTS > 0){
            DeviceVarying<NUM_VARYING_ELEMENTS> dstVarying;
            dstVarying.clear();

            for(int j=0; j<n; ++j){
                int index = F0_IT[h+j];
                dst.addWithWeight(&vertex[index], weight);
                dstVarying.addVaryingWithWeight(&varyings[index], weight);
            }
            vertex[offset + i - tableOffset] = dst;
            varyings[offset + i - tableOffset] = dstVarying;
        }else{
            for(int j=0; j<n; ++j){
                int index = F0_IT[h+j];
                dst.addWithWeight(&vertex[index], weight);
            }
            vertex[offset + i - tableOffset] = dst;
        }
    }
}

__global__ void
computeFace(float *fVertex, int numVertexElements, float *fVaryings, int numVaryingElements,
            int *F0_IT, int *F0_ITa, int offset, int tableOffset, int start, int end)
{
    for(int i = start + tableOffset +threadIdx.x + blockIdx.x*blockDim.x; i < end + tableOffset; i += blockDim.x * gridDim.x){
        int h = F0_ITa[2*i];
        int n = F0_ITa[2*i+1];
        float weight = 1.0f/n;

        // XXX: can we use local stack like alloca?
        float *dstVertex = fVertex + (i+offset-tableOffset)*numVertexElements;
        clear(dstVertex, numVertexElements);
        float *dstVarying = fVaryings + (i+offset-tableOffset)*numVaryingElements;
        clear(dstVarying, numVaryingElements);

        for(int j=0; j<n; ++j){
            int index = F0_IT[h+j];
            addWithWeight(dstVertex, fVertex + index*numVertexElements, weight, numVertexElements);
            addVaryingWithWeight(dstVarying, fVaryings + index*numVaryingElements, weight, numVaryingElements);
        }
    }
}

template <int NUM_USER_VERTEX_ELEMENTS, int NUM_VARYING_ELEMENTS> __global__ void
computeEdge(float *fVertex, float *fVaryings, int *E0_IT, float *E0_S, int offset, int tableOffset, int start, int end)
{
    DeviceVertex<NUM_USER_VERTEX_ELEMENTS> *vertex = (DeviceVertex<NUM_USER_VERTEX_ELEMENTS>*)fVertex;
    DeviceVarying<NUM_VARYING_ELEMENTS> *varyings = (DeviceVarying<NUM_VARYING_ELEMENTS>*)fVaryings;
    for(int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x; i < end + tableOffset; i+= blockDim.x * gridDim.x){
        int eidx0 = E0_IT[4*i+0];
        int eidx1 = E0_IT[4*i+1];
        int eidx2 = E0_IT[4*i+2];
        int eidx3 = E0_IT[4*i+3];

        float vertWeight = E0_S[i*2+0];

        // Fully sharp edge : vertWeight = 0.5f;
        DeviceVertex<NUM_USER_VERTEX_ELEMENTS> dst;
        dst.clear();

        dst.addWithWeight(&vertex[eidx0], vertWeight);
        dst.addWithWeight(&vertex[eidx1], vertWeight);

        if(eidx2 > -1){
            float faceWeight = E0_S[i*2+1];

            dst.addWithWeight(&vertex[eidx2], faceWeight);
            dst.addWithWeight(&vertex[eidx3], faceWeight);
        }
        vertex[offset+i-tableOffset] = dst;

        if(NUM_VARYING_ELEMENTS > 0){
            DeviceVarying<NUM_VARYING_ELEMENTS> dstVarying;
            dstVarying.clear();
            dstVarying.addVaryingWithWeight(&varyings[eidx0], 0.5f);
            dstVarying.addVaryingWithWeight(&varyings[eidx1], 0.5f);
            varyings[offset+i-tableOffset] = dstVarying;
        }
    }
}

__global__ void
computeEdge(float *fVertex, int numVertexElements, float *fVarying, int numVaryingElements,
            int *E0_IT, float *E0_S, int offset, int tableOffset, int start, int end)
{
    for(int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x; i < end + tableOffset; i+= blockDim.x * gridDim.x){
        int eidx0 = E0_IT[4*i+0];
        int eidx1 = E0_IT[4*i+1];
        int eidx2 = E0_IT[4*i+2];
        int eidx3 = E0_IT[4*i+3];

        float vertWeight = E0_S[i*2+0];

        // Fully sharp edge : vertWeight = 0.5f;
        float *dstVertex = fVertex + (i+offset-tableOffset)*numVertexElements;
        clear(dstVertex, numVertexElements);

        addWithWeight(dstVertex, fVertex + eidx0*numVertexElements, vertWeight, numVertexElements);
        addWithWeight(dstVertex, fVertex + eidx1*numVertexElements, vertWeight, numVertexElements);

        if(eidx2 > -1){
            float faceWeight = E0_S[i*2+1];

            addWithWeight(dstVertex, fVertex + eidx2*numVertexElements, faceWeight, numVertexElements);
            addWithWeight(dstVertex, fVertex + eidx3*numVertexElements, faceWeight, numVertexElements);
        }

        if(numVaryingElements > 0){
            float *dstVarying = fVarying + (i+offset-tableOffset)*numVaryingElements;
            clear(dstVarying, numVaryingElements);

            addVaryingWithWeight(dstVarying, fVarying + eidx0*numVaryingElements, 0.5f, numVaryingElements);
            addVaryingWithWeight(dstVarying, fVarying + eidx1*numVaryingElements, 0.5f, numVaryingElements);
        }
    }
}

template <int NUM_USER_VERTEX_ELEMENTS, int NUM_VARYING_ELEMENTS> __global__ void
computeVertexA(float *fVertex, float *fVaryings, int *V0_ITa, float *V0_S, int offset, int tableOffset, int start, int end, int pass)
{
    DeviceVertex<NUM_USER_VERTEX_ELEMENTS> *vertex = (DeviceVertex<NUM_USER_VERTEX_ELEMENTS>*)fVertex;
    DeviceVarying<NUM_VARYING_ELEMENTS> *varyings = (DeviceVarying<NUM_VARYING_ELEMENTS>*)fVaryings;
    for(int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x; i < end+tableOffset; i += blockDim.x * gridDim.x){
        int n     = V0_ITa[5*i+1];
        int p     = V0_ITa[5*i+2];
        int eidx0 = V0_ITa[5*i+3];
        int eidx1 = V0_ITa[5*i+4];

        float weight = (pass==1) ? V0_S[i] : 1.0f - V0_S[i];

        // In the case of fractional weight, the weight must be inverted since
        // the value is shared with the k_Smooth kernel (statistically the
        // k_Smooth kernel runs much more often than this one)
        if (weight>0.0f && weight<1.0f && n > 0)
            weight=1.0f-weight;

        DeviceVertex<NUM_USER_VERTEX_ELEMENTS> dst;
        if (not pass) {
            dst.clear();
        } else {
            dst = vertex[i+offset-tableOffset];
        }

        if (eidx0==-1 || (pass==0 && (n==-1)) ) {
            dst.addWithWeight(&vertex[p], weight);
        } else {
            dst.addWithWeight(&vertex[p], weight * 0.75f);
            dst.addWithWeight(&vertex[eidx0], weight * 0.125f);
            dst.addWithWeight(&vertex[eidx1], weight * 0.125f);
        }
        vertex[i+offset-tableOffset] = dst;

        if(NUM_VARYING_ELEMENTS > 0){
            if(not pass){
                DeviceVarying<NUM_VARYING_ELEMENTS> dstVarying;
                dstVarying.clear();
                dstVarying.addVaryingWithWeight(&varyings[p], 1.0f);
                varyings[i+offset-tableOffset] = dstVarying;
            }
        }
    }
}

__global__ void
computeVertexA(float *fVertex, int numVertexElements, float *fVaryings, int numVaryingElements,
               int *V0_ITa, float *V0_S, int offset, int tableOffset, int start, int end, int pass)
{
    for(int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x; i < end + tableOffset; i += blockDim.x * gridDim.x){
        int n     = V0_ITa[5*i+1];
        int p     = V0_ITa[5*i+2];
        int eidx0 = V0_ITa[5*i+3];
        int eidx1 = V0_ITa[5*i+4];

        float weight = (pass==1) ? V0_S[i] : 1.0f - V0_S[i];

        // In the case of fractional weight, the weight must be inverted since
        // the value is shared with the k_Smooth kernel (statistically the
        // k_Smooth kernel runs much more often than this one)
        if (weight>0.0f && weight<1.0f && n > 0)
            weight=1.0f-weight;

        float *dstVertex = fVertex + (i+offset-tableOffset)*numVertexElements;
        if (not pass) {
            clear(dstVertex, numVertexElements);
        }

        if (eidx0==-1 || (pass==0 && (n==-1)) ) {
            addWithWeight(dstVertex, fVertex + p*numVertexElements, weight, numVertexElements);
        } else {
            addWithWeight(dstVertex, fVertex + p*numVertexElements, weight*0.75f, numVertexElements);
            addWithWeight(dstVertex, fVertex + eidx0*numVertexElements, weight*0.125f, numVertexElements);
            addWithWeight(dstVertex, fVertex + eidx1*numVertexElements, weight*0.125f, numVertexElements);
        }

        if(numVaryingElements > 0){
            if(not pass){
                float *dstVarying = fVaryings + (i+offset-tableOffset)*numVaryingElements;
                clear(dstVarying, numVaryingElements);
                addVaryingWithWeight(dstVarying, fVaryings + p*numVaryingElements, 1.0f, numVaryingElements);
            }
        }
    }

}


//texture <int, 1> texV0_IT;

template <int NUM_USER_VERTEX_ELEMENTS, int NUM_VARYING_ELEMENTS> __global__ void
computeVertexB(float *fVertex, float *fVaryings,
               const int *V0_ITa, const int *V0_IT, const float *V0_S, int offset, int tableOffset, int start, int end)
{
    DeviceVertex<NUM_USER_VERTEX_ELEMENTS> *vertex = (DeviceVertex<NUM_USER_VERTEX_ELEMENTS>*)fVertex;
    DeviceVarying<NUM_VARYING_ELEMENTS> *varyings = (DeviceVarying<NUM_VARYING_ELEMENTS>*)fVaryings;
    for(int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x; i < end + tableOffset; i += blockDim.x * gridDim.x){
        int h = V0_ITa[5*i];
        int n = V0_ITa[5*i+1];
        int p = V0_ITa[5*i+2];

        float weight = V0_S[i];
        float wp = 1.0f/float(n*n);
        float wv = (n-2.0f) * n * wp;

        DeviceVertex<NUM_USER_VERTEX_ELEMENTS> dst;
        dst.clear();
        dst.addWithWeight(&vertex[p], weight * wv);

        for(int j = 0; j < n; ++j){
            dst.addWithWeight(&vertex[V0_IT[h+j*2]], weight * wp);
            dst.addWithWeight(&vertex[V0_IT[h+j*2+1]], weight * wp);
//            int idx0 = tex1Dfetch(texV0_IT, h+j*2);
//            int idx1 = tex1Dfetch(texV0_IT, h+j*2+1);
//            dst.addWithWeight(&vertex[idx0], weight * wp);
//            dst.addWithWeight(&vertex[idx1], weight * wp);
        }
        vertex[i+offset-tableOffset] = dst;

        if(NUM_VARYING_ELEMENTS > 0){
            DeviceVarying<NUM_VARYING_ELEMENTS> dstVarying;
            dstVarying.clear();
            dstVarying.addVaryingWithWeight(&varyings[p], 1.0f);
            varyings[i+offset-tableOffset] = dstVarying;
        }
    }
}

__global__ void
computeVertexB(float *fVertex, int numVertexElements, float *fVaryings, int numVaryingElements,
               const int *V0_ITa, const int *V0_IT, const float *V0_S, int offset, int tableOffset, int start, int end)
{
    for(int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x; i < end + tableOffset; i += blockDim.x * gridDim.x){
        int h = V0_ITa[5*i];
        int n = V0_ITa[5*i+1];
        int p = V0_ITa[5*i+2];

        float weight = V0_S[i];
        float wp = 1.0f/float(n*n);
        float wv = (n-2.0f) * n * wp;

        float *dstVertex = fVertex + (i+offset-tableOffset)*numVertexElements;
        clear(dstVertex, numVertexElements);
        addWithWeight(dstVertex, fVertex + p*numVertexElements, weight*wv, numVertexElements);

        for(int j = 0; j < n; ++j){
            addWithWeight(dstVertex, fVertex + V0_IT[h+j*2]*numVertexElements, weight*wp, numVertexElements);
            addWithWeight(dstVertex, fVertex + V0_IT[h+j*2+1]*numVertexElements, weight*wp, numVertexElements);
        }

        if(numVaryingElements > 0){
            float *dstVarying = fVaryings + (i+offset-tableOffset)*numVaryingElements;
            clear(dstVarying, numVaryingElements);
            addVaryingWithWeight(dstVarying, fVaryings + p*numVaryingElements, 1.0f, numVaryingElements);
        }
    }
}


// --------------------------------------------------------------------------------------------

template <int NUM_USER_VERTEX_ELEMENTS, int NUM_VARYING_ELEMENTS> __global__ void
computeLoopVertexB(float *fVertex, float *fVaryings, int *V0_ITa, int *V0_IT, float *V0_S, int offset, int tableOffset, int start, int end)
{
    DeviceVertex<NUM_USER_VERTEX_ELEMENTS> *vertex = (DeviceVertex<NUM_USER_VERTEX_ELEMENTS>*)fVertex;
    DeviceVarying<NUM_VARYING_ELEMENTS> *varyings = (DeviceVarying<NUM_VARYING_ELEMENTS>*)fVaryings;
    for(int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x; i < end + tableOffset; i += blockDim.x * gridDim.x){
        int h = V0_ITa[5*i];
        int n = V0_ITa[5*i+1];
        int p = V0_ITa[5*i+2];

        float weight = V0_S[i];
        float wp = 1.0f/float(n);
        float beta = 0.25f * __cosf(float(M_PI) * 2.0f * wp) + 0.375f;
        beta = beta * beta;
        beta = (0.625f - beta) * wp;

        DeviceVertex<NUM_USER_VERTEX_ELEMENTS> dst;
        dst.clear();

        dst.addWithWeight(&vertex[p], weight * (1.0f - (beta * n)));

        for(int j = 0; j < n; ++j){
            dst.addWithWeight(&vertex[V0_IT[h+j]], weight * beta);
        }
        vertex[i+offset-tableOffset] = dst;

        if(NUM_VARYING_ELEMENTS > 0){
            DeviceVarying<NUM_VARYING_ELEMENTS> dstVarying;
            dstVarying.clear();
            dstVarying.addVaryingWithWeight(&varyings[p], 1.0f);
            varyings[i+offset-tableOffset] = dstVarying;
        }
    }
}

__global__ void
computeLoopVertexB(float *fVertex, int numVertexElements, float *fVaryings, int numVaryingElements,
                   const int *V0_ITa, const int *V0_IT, const float *V0_S, int offset, int tableOffset, int start, int end)
{
    for(int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x; i < end + tableOffset; i += blockDim.x * gridDim.x){
        int h = V0_ITa[5*i];
        int n = V0_ITa[5*i+1];
        int p = V0_ITa[5*i+2];

        float weight = V0_S[i];
        float wp = 1.0f/float(n);
        float beta = 0.25f * __cosf(float(M_PI) * 2.0f * wp) + 0.375f;
        beta = beta * beta;
        beta = (0.625f - beta) * wp;

        float *dstVertex = fVertex + (i+offset-tableOffset)*numVertexElements;
        clear(dstVertex, numVertexElements);
        addWithWeight(dstVertex, fVertex + p*numVertexElements, weight*(1.0f-(beta*n)), numVertexElements);

        for(int j = 0; j < n; ++j){
            addWithWeight(dstVertex, fVertex + V0_IT[h+j]*numVertexElements, weight*beta, numVertexElements);
        }

        if(numVaryingElements > 0){
            float *dstVarying = fVaryings + (i+offset-tableOffset)*numVaryingElements;
            clear(dstVarying, numVaryingElements);
            addVaryingWithWeight(dstVarying, fVaryings + p*numVaryingElements, 1.0f, numVaryingElements);
        }
    }
}

// --------------------------------------------------------------------------------------------

template <int NUM_USER_VERTEX_ELEMENTS, int NUM_VARYING_ELEMENTS> __global__ void
computeBilinearEdge(float *fVertex, float *fVaryings, int *E0_IT, int offset, int tableOffset, int start, int end)
{
    DeviceVertex<NUM_USER_VERTEX_ELEMENTS> *vertex = (DeviceVertex<NUM_USER_VERTEX_ELEMENTS>*)fVertex;
    DeviceVarying<NUM_VARYING_ELEMENTS> *varyings = (DeviceVarying<NUM_VARYING_ELEMENTS>*)fVaryings;
    for(int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x; i < end + tableOffset; i+= blockDim.x * gridDim.x){
        int eidx0 = E0_IT[2*i+0];
        int eidx1 = E0_IT[2*i+1];

        DeviceVertex<NUM_USER_VERTEX_ELEMENTS> dst;
        dst.clear();

        dst.addWithWeight(&vertex[eidx0], 0.5f);
        dst.addWithWeight(&vertex[eidx1], 0.5f);

        vertex[offset+i-tableOffset] = dst;

        if(NUM_VARYING_ELEMENTS > 0){
            DeviceVarying<NUM_VARYING_ELEMENTS> dstVarying;
            dstVarying.clear();
            dstVarying.addVaryingWithWeight(&varyings[eidx0], 0.5f);
            dstVarying.addVaryingWithWeight(&varyings[eidx1], 0.5f);
            varyings[offset+i-tableOffset] = dstVarying;
        }
    }
}

__global__ void
computeBilinearEdge(float *fVertex, int numVertexElements, float *fVarying, int numVaryingElements,
                    int *E0_IT, int offset, int tableOffset, int start, int end)
{
    for(int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x; i < end + tableOffset; i+= blockDim.x * gridDim.x){
        int eidx0 = E0_IT[2*i+0];
        int eidx1 = E0_IT[2*i+1];

        float *dstVertex = fVertex + (i+offset-tableOffset)*numVertexElements;
        clear(dstVertex, numVertexElements);

        addWithWeight(dstVertex, fVertex + eidx0*numVertexElements, 0.5f, numVertexElements);
        addWithWeight(dstVertex, fVertex + eidx1*numVertexElements, 0.5f, numVertexElements);

        if(numVaryingElements > 0){
            float *dstVarying = fVarying + (i+offset-tableOffset)*numVaryingElements;
            clear(dstVarying, numVaryingElements);

            addVaryingWithWeight(dstVarying, fVarying + eidx0*numVaryingElements, 0.5f, numVaryingElements);
            addVaryingWithWeight(dstVarying, fVarying + eidx1*numVaryingElements, 0.5f, numVaryingElements);
        }
    }
}

template <int NUM_USER_VERTEX_ELEMENTS, int NUM_VARYING_ELEMENTS> __global__ void
computeBilinearVertex(float *fVertex, float *fVaryings, int *V0_ITa, int offset, int tableOffset, int start, int end)
{
    DeviceVertex<NUM_USER_VERTEX_ELEMENTS> *vertex = (DeviceVertex<NUM_USER_VERTEX_ELEMENTS>*)fVertex;
    DeviceVarying<NUM_VARYING_ELEMENTS> *varyings = (DeviceVarying<NUM_VARYING_ELEMENTS>*)fVaryings;
    for(int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x; i < end + tableOffset; i += blockDim.x * gridDim.x){
        int p = V0_ITa[i];

        DeviceVertex<NUM_USER_VERTEX_ELEMENTS> dst;
        dst.clear();

        dst.addWithWeight(&vertex[p], 1.0f);
        vertex[i+offset-tableOffset] = dst;

        if(NUM_VARYING_ELEMENTS > 0){
            DeviceVarying<NUM_VARYING_ELEMENTS> dstVarying;
            dstVarying.clear();
            dstVarying.addVaryingWithWeight(&varyings[p], 1.0f);
            varyings[i+offset-tableOffset] = dstVarying;
        }
    }
}

__global__ void
computeBilinearVertex(float *fVertex, int numVertexElements, float *fVaryings, int numVaryingElements,
                      const int *V0_ITa, int offset, int tableOffset, int start, int end)
{
    for(int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x; i < end + tableOffset; i += blockDim.x * gridDim.x){
        int p = V0_ITa[i];

        float *dstVertex = fVertex + (i+offset-tableOffset)*numVertexElements;
        clear(dstVertex, numVertexElements);
        addWithWeight(dstVertex, fVertex + p*numVertexElements, 1.0f, numVertexElements);

        if(numVaryingElements > 0){
            float *dstVarying = fVaryings + (i+offset-tableOffset)*numVaryingElements;
            clear(dstVarying, numVaryingElements);
            addVaryingWithWeight(dstVarying, fVaryings + p*numVaryingElements, 1.0f, numVaryingElements);
        }
    }
}

// --------------------------------------------------------------------------------------------

__global__ void
editVertexAdd(float *fVertex, int numVertexElements, int primVarOffset, int primVarWidth,
              int vertexOffset, int tableOffset, int start, int end,
              const int *editIndices, const float *editValues)
{
    for(int i = start + tableOffset + threadIdx.x + blockIdx.x*blockDim.x;
        i < end + tableOffset;
        i += blockDim.x * gridDim.x) {

        float *dstVertex = fVertex + (editIndices[i] + vertexOffset) * numVertexElements + primVarOffset;

        for(int j = 0; j < primVarWidth; j++) {
            *dstVertex++ += editValues[i*primVarWidth + j];
        }
    }
}

// --------------------------------------------------------------------------------------------

#include "../version.h"

// XXX: this macro usage is tentative. Since cuda kernel can't be dynamically configured,
// still trying to find better way to have optimized kernel..

#define OPT_KERNEL(NUM_USER_VERTEX_ELEMENTS, NUM_VARYING_ELEMENTS, KERNEL, X, Y, ARG) \
    if(numUserVertexElements == NUM_USER_VERTEX_ELEMENTS && \
       numVaryingElements == NUM_VARYING_ELEMENTS) \
       { KERNEL<NUM_USER_VERTEX_ELEMENTS, NUM_VARYING_ELEMENTS><<<X,Y>>>ARG; \
         return;  }

extern "C" {

void OsdCudaComputeFace(float *vertex, float *varying,
                        int numUserVertexElements, int numVaryingElements,
                        int *F_IT, int *F_ITa, int offset, int tableOffset, int start, int end)
{
    //computeFace<3, 0><<<512,32>>>(vertex, varying, F_IT, F_ITa, offset, start, end);
    OPT_KERNEL(0, 0, computeFace, 512, 32, (vertex, varying, F_IT, F_ITa, offset, tableOffset, start, end));
    OPT_KERNEL(0, 3, computeFace, 512, 32, (vertex, varying, F_IT, F_ITa, offset, tableOffset, start, end));
    OPT_KERNEL(3, 0, computeFace, 512, 32, (vertex, varying, F_IT, F_ITa, offset, tableOffset, start, end));
    OPT_KERNEL(3, 3, computeFace, 512, 32, (vertex, varying, F_IT, F_ITa, offset, tableOffset, start, end));

    // fallback kernel (slow)
    computeFace<<<512, 32>>>(vertex, 3+numUserVertexElements, varying, numVaryingElements,
                             F_IT, F_ITa, offset, tableOffset, start, end);
}

void OsdCudaComputeEdge(float *vertex, float *varying,
                        int numUserVertexElements, int numVaryingElements,
                        int *E_IT, float *E_W, int offset, int tableOffset, int start, int end)
{
    //computeEdge<0, 3><<<512,32>>>(vertex, varying, E_IT, E_W, offset, start, end);
    OPT_KERNEL(0, 0, computeEdge, 512, 32, (vertex, varying, E_IT, E_W, offset, tableOffset, start, end));
    OPT_KERNEL(0, 3, computeEdge, 512, 32, (vertex, varying, E_IT, E_W, offset, tableOffset, start, end));
    OPT_KERNEL(3, 0, computeEdge, 512, 32, (vertex, varying, E_IT, E_W, offset, tableOffset, start, end));
    OPT_KERNEL(3, 3, computeEdge, 512, 32, (vertex, varying, E_IT, E_W, offset, tableOffset, start, end));

    computeEdge<<<512, 32>>>(vertex, 3+numUserVertexElements, varying, numVaryingElements,
                             E_IT, E_W, offset, tableOffset, start, end);
}

void OsdCudaComputeVertexA(float *vertex, float *varying,
                           int numUserVertexElements, int numVaryingElements,
                           int *V_ITa, float *V_W, int offset, int tableOffset, int start, int end, int pass)
{
//    computeVertexA<0, 3><<<512,32>>>(vertex, varying, V_ITa, V_W, offset, start, end, pass);
    OPT_KERNEL(0, 0, computeVertexA, 512, 32, (vertex, varying, V_ITa, V_W, offset, tableOffset, start, end, pass));
    OPT_KERNEL(0, 3, computeVertexA, 512, 32, (vertex, varying, V_ITa, V_W, offset, tableOffset, start, end, pass));
    OPT_KERNEL(3, 0, computeVertexA, 512, 32, (vertex, varying, V_ITa, V_W, offset, tableOffset, start, end, pass));
    OPT_KERNEL(3, 3, computeVertexA, 512, 32, (vertex, varying, V_ITa, V_W, offset, tableOffset, start, end, pass));

    computeVertexA<<<512, 32>>>(vertex, 3+numUserVertexElements, varying, numVaryingElements,
                                V_ITa, V_W, offset, tableOffset, start, end, pass);
}

void OsdCudaComputeVertexB(float *vertex, float *varying,
                           int numUserVertexElements, int numVaryingElements,
                           int *V_ITa, int *V_IT, float *V_W, int offset, int tableOffset, int start, int end)
{
//    computeVertexB<0, 3><<<512,32>>>(vertex, varying, V_ITa, V_IT, V_W, offset, start, end);
    OPT_KERNEL(0, 0, computeVertexB, 512, 32, (vertex, varying, V_ITa, V_IT, V_W, offset, tableOffset, start, end));
    OPT_KERNEL(0, 3, computeVertexB, 512, 32, (vertex, varying, V_ITa, V_IT, V_W, offset, tableOffset, start, end));
    OPT_KERNEL(3, 0, computeVertexB, 512, 32, (vertex, varying, V_ITa, V_IT, V_W, offset, tableOffset, start, end));
    OPT_KERNEL(3, 3, computeVertexB, 512, 32, (vertex, varying, V_ITa, V_IT, V_W, offset, tableOffset, start, end));

    computeVertexB<<<512, 32>>>(vertex, 3+numUserVertexElements, varying, numVaryingElements,
                                V_ITa, V_IT, V_W, offset, tableOffset, start, end);
}

void OsdCudaComputeLoopVertexB(float *vertex, float *varying,
                               int numUserVertexElements, int numVaryingElements,
                               int *V_ITa, int *V_IT, float *V_W, int offset, int tableOffset, int start, int end)
{
//    computeLoopVertexB<0, 3><<<512,32>>>(vertex, varying, V_ITa, V_IT, V_W, offset, start, end);
    OPT_KERNEL(0, 0, computeLoopVertexB, 512, 32, (vertex, varying, V_ITa, V_IT, V_W, offset, tableOffset, start, end));
    OPT_KERNEL(0, 3, computeLoopVertexB, 512, 32, (vertex, varying, V_ITa, V_IT, V_W, offset, tableOffset, start, end));
    OPT_KERNEL(3, 0, computeLoopVertexB, 512, 32, (vertex, varying, V_ITa, V_IT, V_W, offset, tableOffset, start, end));
    OPT_KERNEL(3, 3, computeLoopVertexB, 512, 32, (vertex, varying, V_ITa, V_IT, V_W, offset, tableOffset, start, end));

    computeLoopVertexB<<<512, 32>>>(vertex, 3+numUserVertexElements, varying, numVaryingElements,
                                    V_ITa, V_IT, V_W, offset, tableOffset, start, end);
}

void OsdCudaComputeBilinearEdge(float *vertex, float *varying,
                                int numUserVertexElements, int numVaryingElements,
                                int *E_IT, int offset, int tableOffset, int start, int end)
{
    //computeBilinearEdge<0, 3><<<512,32>>>(vertex, varying, E_IT, offset, start, end);
    OPT_KERNEL(0, 0, computeBilinearEdge, 512, 32, (vertex, varying, E_IT, offset, tableOffset, start, end));
    OPT_KERNEL(0, 3, computeBilinearEdge, 512, 32, (vertex, varying, E_IT, offset, tableOffset, start, end));
    OPT_KERNEL(3, 0, computeBilinearEdge, 512, 32, (vertex, varying, E_IT, offset, tableOffset, start, end));
    OPT_KERNEL(3, 3, computeBilinearEdge, 512, 32, (vertex, varying, E_IT, offset, tableOffset, start, end));

    computeBilinearEdge<<<512, 32>>>(vertex, 3+numUserVertexElements, varying, numVaryingElements,
                                     E_IT, offset, tableOffset, start, end);
}

void OsdCudaComputeBilinearVertex(float *vertex, float *varying,
                                  int numUserVertexElements, int numVaryingElements,
                                  int *V_ITa, int offset, int tableOffset, int start, int end)
{
//    computeBilinearVertex<0, 3><<<512,32>>>(vertex, varying, V_ITa, offset, start, end);
    OPT_KERNEL(0, 0, computeBilinearVertex, 512, 32, (vertex, varying, V_ITa, offset, tableOffset, start, end));
    OPT_KERNEL(0, 3, computeBilinearVertex, 512, 32, (vertex, varying, V_ITa, offset, tableOffset, start, end));
    OPT_KERNEL(3, 0, computeBilinearVertex, 512, 32, (vertex, varying, V_ITa, offset, tableOffset, start, end));
    OPT_KERNEL(3, 3, computeBilinearVertex, 512, 32, (vertex, varying, V_ITa, offset, tableOffset, start, end));

    computeBilinearVertex<<<512, 32>>>(vertex, 3+numUserVertexElements, varying, numVaryingElements,
                                       V_ITa, offset, tableOffset, start, end);
}

void OsdCudaEditVertexAdd(float *vertex, int numUserVertexElements,
                          int primVarOffset, int primVarWidth,
                          int vertexOffset, int tableOffset,
                          int start, int end, int *editIndices, float *editValues)
{
    editVertexAdd<<<512, 32>>>(vertex, 3+numUserVertexElements, primVarOffset, primVarWidth,
                               vertexOffset, tableOffset, start, end,
                               editIndices, editValues);
}

}  /* extern "C" */
